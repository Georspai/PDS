#include "hip/hip_runtime.h"
//Parallel and Distributed Systems
//
//Spaias Georgios
//AEM: 8910

#include "hip/hip_runtime.h"
#include ""


#include "stdio.h"
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <time.h>
#include <math.h>
#include "Utils.cuh"


void sequential_Ising(int8_t*** G, int8_t*** G_trans, int n, int k);

void calculate_grid(int* numOfBlocks, int* numOfThreads, int pointsPerThread, int lattice_size);

__host__ void Ising_comp(int8_t** G, int8_t** G_trans, int* size, int* steps, int* numofBlocks, int* numofThreads, int* pointsPerThread, int tile_size);

__host__ void swap_dptr(void** x, void** y);

__host__ __device__ int8_t sgn(int8_t x);

__global__ void compute_State(int8_t* a, int8_t* b, int* size, int* dev_pointsPerThread);


int main(int argc, char* argv[]) {

    //n:Lattice Dimension & k:Number of Iterations
    int n =80, k =1, cnvrg = 0;
    int* dev_n;
    int* dev_cnvrg;

    int numOfBlocks = 1;
    int numOfThreads = 1;
    int pointsPerThread = 5;

    dev_n = (int*)malloc(sizeof(int));
    dev_cnvrg = (int*)malloc(sizeof(int));

    int8_t** lattice;
    int8_t* dev_lattice = (int8_t*)malloc(n * n * sizeof(int8_t));
    int8_t* dev_lattice_trans = (int8_t*)malloc(n * n * sizeof(int8_t));
    int8_t** lat_trans;

    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    FILE* f_ptr;
    if (EVALUATION_MODE)
    {

        f_ptr = fopen("eval.bin", "rb");
        if (f_ptr == NULL)
        {
            fprintf(stderr, "main: Failed to open eval.bin .\n");
            exit(1);
        }
        fread(&n, sizeof(int), 1, f_ptr);
        fread(&k, sizeof(int), 1, f_ptr);
        printf("Generated Lattice with n=%d and k=%d\n", n, k);
        fclose(f_ptr);
    }

    //Allocate memory for the host lattice
    lattice = lattice_init(n);
    lat_trans = lattice_init(n);

    //Initialize lattice with a random starting state
    start_state(lattice, n);
    printf("GPU ISING COMPUTATION V2\n\n");
    //printLattice(lattice, &n);

    cudaStatus = latticeInitCuda(&lattice, &dev_lattice, &dev_lattice_trans, &n, &cnvrg, &dev_n, &dev_cnvrg);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "latticeInitCuda Failed:\t%s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    hipDeviceSynchronize();

    calculate_grid(&numOfBlocks, &numOfThreads, pointsPerThread, n);
    int tile_size = (((numOfThreads*pointsPerThread)/n+1)*n+2*n) * sizeof(int8_t);
    hipEventRecord(start);

    Ising_comp(&dev_lattice, &dev_lattice_trans, dev_n, &k, &numOfBlocks, &numOfThreads, &pointsPerThread, tile_size);

    hipDeviceSynchronize();

    hipEventRecord(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("1~GPU Ising Computation: Done\n");
    //printf("lat Address %p G Address %p |lat_trans Address %p  G_trans Address %p\n", &dev_lattice, dev_lattice, &dev_lattice_trans, dev_lattice_trans);


    sequential_Ising(&lattice, &lat_trans, n, k);
    printf("2~Sequential Ising Computation: Done\n");


    for (size_t i = 0; i < n; i++)
    {
        cudaStatus = hipMemcpy(*(lattice + i), (dev_lattice + n * i), n * sizeof(int8_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed1!");
            return cudaStatus;
        }
    }
    //printLattice(lattice, &n);
    int flag = 1;
    for (int i = 0; i < n; i++)
    {
         for (int j = 0; j < n; j++)
         {
             if (lattice[i][j] != lat_trans[i][j])
             {
                 //printf("\nInconsistent outcome between parallel and Sequential computation on element: %d/%d!~~~~~~\n\n ", i * n + j, n * n);
                 flag = 0;
             }
         }
    }
    if (flag) printf("3~GPU and Sequential Ising Computation are consistent\n");
    
    //printLattice(lat_trans, &n);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nIterations(k)\tDim(n)\tMaxThreadNumPerBlock\tNumofBlocks\tnumOfThreads\tpointsPerThread\tElapsed time (ms)\n");
    printf("%d\t%d\t1024\t%d\t%d\t%d\t%f\n", k, n, numOfBlocks, numOfThreads, pointsPerThread, milliseconds);

    hipFree(dev_lattice);
    hipFree(dev_lattice_trans);
    hipFree(dev_n);
    hipFree(dev_cnvrg);

    free(lattice);
    free(lat_trans);


    return 0;
}







__host__ void Ising_comp(int8_t** lat, int8_t** lat_trans, int* size, int* steps, int* numofBlocks, int* numofThreads, int* pointsPerThread ,int tile_size)
{
    hipError_t cudaStatus;
    int k = *steps;
    
    int* dev_pointsperThread = 0;
    cudaStatus = hipMalloc(&dev_pointsperThread, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA MALLOC Failed:\t%s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipMemcpy((void*)dev_pointsperThread, (void*)pointsPerThread, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "MemCopy Failed:\t%s\n", hipGetErrorString(cudaStatus));
    }
    hipDeviceSynchronize();
    for (int i = 0; i < k; i++)
    {
        //hipDeviceSynchronize();
        compute_State <<< *numofBlocks, *numofThreads , tile_size >>> (*lat, *lat_trans, size, dev_pointsperThread);
        fprintf(stderr, "kernel Launch Error:\t%s\n", hipGetErrorString(hipGetLastError()));
        hipDeviceSynchronize();
        printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
        swap_dptr((void**)lat, (void**)lat_trans);
        
        fprintf(stderr, "Sync Error:\t%s\n", hipGetErrorString(hipGetLastError()));

    }
    
    hipFree(dev_pointsperThread);
    fprintf(stderr, "GPU Ising Computation finished with:\t%s\n", hipGetErrorString(hipGetLastError()));
}


__global__ void compute_State(int8_t* G, int8_t* G_trans, int* size, int* pointsPerThread)
{   
    extern __shared__ int8_t shrd_tile[];
    
    int sum,i, j;
    int ppt = *pointsPerThread;
    int n = *size;
    int tile_size = ((((int) blockDim.x )* ppt) / n + 1) * n +2*n;
    int k = tile_size / n;
    int threadIndex = (blockIdx.x * blockDim.x + threadIdx.x) * ppt;
    int index_start = ((threadIndex/n)+1)* n + threadIndex % n;
    int index_end = index_start + blockDim.x * ppt;
    printf("%d Part 0 with TileSize: %d |blockDim %d |blockIdx %d | ppt %d\n", threadIndex, tile_size, blockDim.x, blockIdx.x, ppt);
    for (int k = 0; k < ppt; k++)
    {
        shrd_tile[index_start + k] = G[threadIndex + k];
    }
    printf("Part 1 \n");
    if (threadIdx.x==0)
    {
       
        for (int k = 0; k < (tile_size- index_end); k++)
        {  
            //printf("\n%d\n", (blockDim.x * ppt + i) % (n * n));
            shrd_tile[index_end+k] =  G[(blockDim.x * ppt  + k) % (n * n)];
        }
    }
    //printf("Part 2 \n");
    if (threadIdx.x == 0)
    {   
        for (int k = 0; k < index_start ; k++)
        {
            shrd_tile[k] = G[(threadIndex - index_start+ n*n+ k) % (n * n)];
        }
    }
    //printf("Part 3 \n");
    __syncthreads();

    //if (threadIdx.x == 0)
    //{
    //    printf("TILE_SIZE: %d , index_start: %d , index_end: %d\n", tile_size, index_start, index_end);
    //    for (int i = 0; i <tile_size; i++)
    //    {
    //        if (i % n == 0) { printf("\n"); }
    //        printf("%d\t", shrd_tile[i]);
    //    }
    //}
    sum = 0;
    for (int index =0 ; index < ppt; index++)
    {
        i = (index_start + index) / n;
        j = (index_start + index) % n;

        if (threadIndex + index < (n*n))
        {
            sum = shrd_tile[i * n + j];
            //neighbor 1 -->(i,j+1)
            sum += shrd_tile[(i * n + (j + 1) % n)];
            //neighbor 2 -->(i,j-1)
            sum += shrd_tile[(i * n + (j - 1 + n) % n) ];
            //neighbor 3 -->(i-1,j)
       
            sum += shrd_tile[(((i - 1 + n) % n) * n + j) ];
            //neighbor 4 -->(i+1,j)
            
            sum += shrd_tile[(((i + 1) % n) * n + j) ];

            G_trans[(threadIndex+index)] = sgn(sum);
        }
    }
    
}


__forceinline__ __host__ __device__ int8_t sgn(int8_t x)
{
    x = (int8_t)(x > 0) - (int8_t)(x < 0);
    return x;
}

__host__ void swap_dptr(void** x, void** y)
{
    void* temp = *x;
    *x = *y;
    *y = temp;
}




void calculate_grid(int* numOfBlocks, int* numOfThreads, int pointsPerThread,  int lattice_size) {
    hipDeviceProp_t cudaProperties;
    hipGetDeviceProperties(&cudaProperties, 0);
    int maxThreadsPerBlock = cudaProperties.maxThreadsPerBlock;
    int lat_points = lattice_size * lattice_size;
    int pointsPerThreadSquared = pointsPerThread ;
    int block_num = 1;
    int thread_num = 1;

    block_num = lat_points / (pointsPerThreadSquared * maxThreadsPerBlock) + ((lat_points % (pointsPerThreadSquared * maxThreadsPerBlock)) > 0);
    thread_num = lat_points / (pointsPerThreadSquared * block_num) + ((lat_points % (pointsPerThreadSquared * block_num)) > 0);
    
    int tile_size = (((thread_num * pointsPerThread) / lattice_size + 1) * lattice_size + 2 * lattice_size) * sizeof(int8_t);
    printf("max thread num: %d ||block_num: % d || thread_num : % d|| tile size : %d\n", maxThreadsPerBlock, block_num, thread_num, tile_size);

    *numOfBlocks = block_num;
    *numOfThreads = thread_num;

}


void sequential_Ising(int8_t*** lat, int8_t*** lat_trans, int n, int k) {

    int8_t** G = *lat;
    int8_t** G_trans = *lat_trans;

    int cnvrg = 0;
    int n_i = 0, n_j = 0;
    for (int l = 0; l < k; l++)
    {
        cnvrg = 0;
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                G_trans[i][j] = 0;
                n_i = (i + 1) % n;              //neighbor 1 -->(i+1,j)
                n_j = j;
                G_trans[i][j] += G[n_i][n_j];
                n_i = i;                        //neighbor 2 -->(i,j+1)
                n_j = (j + 1) % n;
                G_trans[i][j] += G[n_i][n_j];
                n_i = (i - 1 + n) % n;          //neighbor 3 -->(i-1,j)
                n_j = j;
                G_trans[i][j] += G[n_i][n_j];
                n_i = i;                        //neighbor 4 -->(i,j-1)
                n_j = (j - 1 + n) % n;
                G_trans[i][j] += G[n_i][n_j];
                G_trans[i][j] += G[i][j];
                G_trans[i][j] = sgn(G_trans[i][j]);
                cnvrg += abs(sgn(G_trans[i][j] - G[i][j]));
            }
        }
        if (cnvrg == 0)
        {
            printf("\nSequential Ising model has converged on step %ld/%d \n", l, k);
            break;
        }
        swap_dptr((void**)&G, (void**)&G_trans);

    }
    swap_dptr((void**)&G, (void**)&G_trans);
    *lat = G;
    *lat_trans = G_trans;
}